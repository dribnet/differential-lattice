
#include <hip/hip_runtime.h>
__global__ void step(
  int n,
  int nz,
  int zone_leap,
  float *xy,
  int *potential,
  int *zone_num,
  int *zone_node,
  float stp,
  float reject_stp,
  float attract_stp,
  float spring_stp,
  float spring_reject_rad,
  float spring_attract_rad,
  int max_capacity,
  float node_rad,
  float max_rad
){
  const int i = blockIdx.x*512 + threadIdx.x;

  if (i>=n) {
    return;
  }

  const int ii = 2*i;
  const int zi = (int) floor(xy[ii]*nz);
  const int zj = (int) floor(xy[ii+1]*nz);
  const int z = zi*nz + zj;

  float sx = 0;
  float sy = 0;
  float dx = 0;
  float dy = 0;
  float dd = 0;

  int jj;
  int aa;
  int zk;

  int edge_count = 0;
  int cand_count = 0;

  bool linked;

  int old = atomicAdd(&zone_num[z], 1);
  zone_node[z*zone_leap+old] = i;

  int proximity[1000];

  __syncthreads();

  for (int a=max(zi-1,0);a<min(zi+2,nz);a++){
    for (int b=max(zj-1,0);b<min(zj+2,nz);b++){
      zk = a*nz+b;
      for (int k=0;k<zone_num[zk];k++){
        jj = 2*zone_node[zk*zone_leap+k];
        dx = xy[ii] - xy[jj];
        dy = xy[ii+1] - xy[jj+1];
        dd = sqrt(dx*dx + dy*dy);
        if (dd<max_rad){
          proximity[cand_count] = jj/2;
          cand_count += 1;
        }
      }
    }
  }

  for (int k=0;k<cand_count;k++){

    jj = 2*proximity[k];

    dx = xy[ii] - xy[jj];
    dy = xy[ii+1] - xy[jj+1];
    dd = sqrt(dx*dx + dy*dy);

    linked = true;
    for (int l=0;l<cand_count;l++){
      aa = 2*proximity[l];
      if (dd>max(
          sqrt(powf(xy[ii] - xy[aa],2.0) + powf(xy[ii+1] - xy[aa+1],2.0)),
          sqrt(powf(xy[jj] - xy[aa],2.0) + powf(xy[jj+1] - xy[aa+1],2.0))
        )
      ){
        linked = false;
        break;
      }
    }

    if (dd>0.0){

      dx /= dd;
      dy /= dd;

      if (linked){
        edge_count += 1;
        if (dd>spring_attract_rad){
          sx += -dx*spring_stp;
          sy += -dy*spring_stp;
        }
        else if(dd<spring_reject_rad){
          sx += dx*spring_stp;
          sy += dy*spring_stp;
        }
      }
      else{ // unlinked
        if (potential[i]>0 && potential[jj/2]>0){
          sx += -dx*attract_stp;
          sy += -dy*attract_stp;
        }
        else{
          sx += dx*reject_stp;
          sy += dy*reject_stp;
        }
      }
    }
  }

  __syncthreads();

  xy[ii] = xy[ii] + sx*stp;
  xy[ii+1] = xy[ii+1] + sy*stp;
  /*potential[i] = cand_count;*/
  if (cand_count<max_capacity){
    potential[i] = cand_count;
  }
  else{
    potential[i] = 0;
  }

}
